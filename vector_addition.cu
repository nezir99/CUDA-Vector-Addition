//write C/C++ code for vector addition (and add cuda)

#include <hip/hip_runtime.h>

#include <stdio.h>
#include <stdlib.h>


__global__ void addVectors(float* a, float* b, float* c, int n) {
    int index = blockIdx.x * blockDim.x + threadIdx.x;
    if (index < n) {
        c[index] = a[index] + b[index];
    }
}


//the main function - necessary for all C programs
int main() {
    int n = 1000000;  // Size of vectors
    size_t size = n * sizeof(float);

    // Allocate host memory
    float *h_a, *h_b, *h_c;
    h_a = (float*)malloc(size);
    h_b = (float*)malloc(size);
    h_c = (float*)malloc(size);

    // Initialize vectors (choose an arbitrary value for each of the two vectors being added)
    for (int i = 0; i < n; i++) {
        h_a[i] = i;
        h_b[i] = i * 2;
    }    
    
    // Allocate device memory
    float *d_a, *d_b, *d_c;
    hipMalloc(&d_a, size);
    hipMalloc(&d_b, size);
    hipMalloc(&d_c, size);

    // Copy input data from host to device
    hipMemcpy(d_a, h_a, size, hipMemcpyHostToDevice);
	hipMemcpy(d_b, h_b, size, hipMemcpyHostToDevice);
    
     // Launch kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n + threadsPerBlock - 1) / threadsPerBlock;
    addVectors<<<blocksPerGrid, threadsPerBlock>>>(d_a, d_b, d_c, n);

    // Copy result back to host
    hipMemcpy(h_c, d_c, size, hipMemcpyDeviceToHost);
    
    // Print first few results
    for (int i = 0; i < 10; i++) {
        printf("%f + %f = %f\n", h_a[i], h_b[i], h_c[i]);
    }
    
    
    // Free memory
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_c);
    free(h_a);
    free(h_b);
    free(h_c);

    return 0;
}